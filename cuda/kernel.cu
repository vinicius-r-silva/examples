#include <iostream>
#include <hip/hip_runtime.h>

// CUDA kernel to add two arrays
__global__ void add_arrays(int *a, int *b, int *c, int size) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // printf("%d", index);
    if (index < size) {
        // printf("%d", index);
        c[index] = a[index] + b[index];
    }
}

int main() {
    const int array_size = 100;
    const int array_bytes = array_size * sizeof(int);

    // Allocate memory on the host (CPU)
    int *h_a = new int[array_size];
    int *h_b = new int[array_size];
    int *h_c = new int[array_size];

    // Initialize arrays on the host
    for (int i = 0; i < array_size; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Allocate memory on the device (GPU)
    int *d_a, *d_b, *d_c;
    hipMalloc(&d_a, array_bytes);
    hipMalloc(&d_b, array_bytes);
    hipMalloc(&d_c, array_bytes);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, array_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, array_bytes, hipMemcpyHostToDevice);

    // Launch the CUDA kernel with 10 blocks of 10 threads each
    add_arrays<<<(array_size + 9) / 10, 10>>>(d_a, d_b, d_c, array_size);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    }

    // Copy the result back from device to host
    hipMemcpy(h_c, d_c, array_bytes, hipMemcpyDeviceToHost);

    // Output the result
    for (int i = 0; i < 10; i++) {  // Show the first 10 results
        std::cout << h_c[i] << " ";
    }
    std::cout << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    // Free host memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
